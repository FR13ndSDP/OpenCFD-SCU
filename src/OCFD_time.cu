#include "hip/hip_runtime.h"
#include "parameters.h"
#include "utility.h"
#include "OCFD_time.h"

#include "cuda_commen.h"
#include "cuda_utility.h"
#include "parameters_d.h"


#ifdef __cplusplus
extern "C"{
#endif

// 旋转源项
// __global__ void source_kernel(cudaSoA du , cudaField rho , cudaField v, cudaField w, cudaField yy, cudaField zz, cudaJobPackage job)
// {
//     // eyes on no-lap region
//     unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
//     unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
//     unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

//     // TODO: define omega elsewhere
//     REAL Omega = 0.002;

//     if( x<job.end.x && y<job.end.y && z<job.end.z){

//         REAL y_ = get_Field_LAP(yy, x+LAP,y+LAP,z+LAP);
//         REAL z_ = get_Field_LAP(zz, x+LAP,y+LAP,z+LAP);
//         REAL rho_ = get_Field_LAP(rho, x+LAP,y+LAP,z+LAP);
//         REAL v_ = get_Field_LAP(v, x+LAP,y+LAP,z+LAP);
//         REAL w_ = get_Field_LAP(w, x+LAP,y+LAP,z+LAP);

//         REAL f_2 = rho_*(Omega*Omega * y_ + 2.0 * Omega * w_);
//         REAL f_3 = rho_*(Omega*Omega * z_ - 2.0 * Omega * v_);
//         REAL f_4 = rho_* Omega*Omega * (v_ * y_  + w_ * z_);

//         get_SoA(du , x,y,z , 2) += f_2;
//         get_SoA(du , x,y,z , 3) += f_3;
//         get_SoA(du , x,y,z , 4) += f_4;

//     }
// }

__global__ void OCFD_time_advance_ker1(cudaSoA f , cudaSoA fn , cudaSoA du , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		get_SoA(f , x,y,z , 0) = get_SoA(fn , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0);
		get_SoA(f , x,y,z , 1) = get_SoA(fn , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1);
		get_SoA(f , x,y,z , 2) = get_SoA(fn , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2);
		get_SoA(f , x,y,z , 3) = get_SoA(fn , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3);
		get_SoA(f , x,y,z , 4) = get_SoA(fn , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4);

	}
}

__global__ void OCFD_time_advance_ker2(cudaSoA f , cudaSoA fn , cudaSoA du , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		REAL tmp1 = 3.0 / 4.0;
		REAL tmp2 = 1.0 / 4.0;

		get_SoA(f , x,y,z , 0) = tmp1*get_SoA(fn , x,y,z , 0) + tmp2*( get_SoA(f , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0) );
		get_SoA(f , x,y,z , 1) = tmp1*get_SoA(fn , x,y,z , 1) + tmp2*( get_SoA(f , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1) );
		get_SoA(f , x,y,z , 2) = tmp1*get_SoA(fn , x,y,z , 2) + tmp2*( get_SoA(f , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2) );
		get_SoA(f , x,y,z , 3) = tmp1*get_SoA(fn , x,y,z , 3) + tmp2*( get_SoA(f , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3) );
		get_SoA(f , x,y,z , 4) = tmp1*get_SoA(fn , x,y,z , 4) + tmp2*( get_SoA(f , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4) );
	}
}

__global__ void OCFD_time_advance_ker3(cudaSoA f , cudaSoA fn , cudaSoA du , cudaSoA f_lap , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		REAL tmp1 = 1.0 / 3.0;
		REAL tmp2 = 2.0 / 3.0;

		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 0) = get_SoA(f , x,y,z , 0) = tmp1*get_SoA(fn , x,y,z , 0) + tmp2*( get_SoA(f , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 1) = get_SoA(f , x,y,z , 1) = tmp1*get_SoA(fn , x,y,z , 1) + tmp2*( get_SoA(f , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 2) = get_SoA(f , x,y,z , 2) = tmp1*get_SoA(fn , x,y,z , 2) + tmp2*( get_SoA(f , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 3) = get_SoA(f , x,y,z , 3) = tmp1*get_SoA(fn , x,y,z , 3) + tmp2*( get_SoA(f , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 4) = get_SoA(f , x,y,z , 4) = tmp1*get_SoA(fn , x,y,z , 4) + tmp2*( get_SoA(f , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4) );
	}
}

void OCFD_time_advance(int KRK)
{
	dim3 griddim , blockdim;
    cal_grid_block_dim(&griddim , &blockdim , BlockDimX , BlockDimY , BlockDimZ , nx,ny,nz);
    cudaJobPackage job( dim3(0,0,0) , dim3(nx,ny,nz) );

	// The RHS source is added here for every RK step
	// CUDA_LAUNCH(( source_kernel<<<griddim , blockdim>>>(*pdu_d, *pd_d, *pv_d, *pw_d, *pAyy_d, *pAzz_d, job)));

	switch (KRK)
	{
		case 1:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker1<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , job) ));
			break;
		}
		case 2:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker2<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , job) ));
			break;
		}
		case 3:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker3<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , *pf_lap_d , job) ));
			break;
		}
	}
}


#ifdef __cplusplus
}
#endif