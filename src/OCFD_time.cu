#include "hip/hip_runtime.h"
#include "parameters.h"
#include "utility.h"
#include "OCFD_time.h"

#include "cuda_commen.h"
#include "cuda_utility.h"
#include "parameters_d.h"


#ifdef __cplusplus
extern "C"{
#endif

__global__ void OCFD_time_advance_ker1(cudaSoA f , cudaSoA fn , cudaSoA du , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		get_SoA(f , x,y,z , 0) = get_SoA(fn , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0);
		get_SoA(f , x,y,z , 1) = get_SoA(fn , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1);
		get_SoA(f , x,y,z , 2) = get_SoA(fn , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2);
		get_SoA(f , x,y,z , 3) = get_SoA(fn , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3);
		get_SoA(f , x,y,z , 4) = get_SoA(fn , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4);

	}
}

__global__ void OCFD_time_advance_ker2(cudaSoA f , cudaSoA fn , cudaSoA du , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		REAL tmp1 = 3.0 / 4.0;
		REAL tmp2 = 1.0 / 4.0;

		get_SoA(f , x,y,z , 0) = tmp1*get_SoA(fn , x,y,z , 0) + tmp2*( get_SoA(f , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0) );
		get_SoA(f , x,y,z , 1) = tmp1*get_SoA(fn , x,y,z , 1) + tmp2*( get_SoA(f , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1) );
		get_SoA(f , x,y,z , 2) = tmp1*get_SoA(fn , x,y,z , 2) + tmp2*( get_SoA(f , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2) );
		get_SoA(f , x,y,z , 3) = tmp1*get_SoA(fn , x,y,z , 3) + tmp2*( get_SoA(f , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3) );
		get_SoA(f , x,y,z , 4) = tmp1*get_SoA(fn , x,y,z , 4) + tmp2*( get_SoA(f , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4) );
	}
}

__global__ void OCFD_time_advance_ker3(cudaSoA f , cudaSoA fn , cudaSoA du , cudaSoA f_lap , cudaJobPackage job)
{
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + job.start.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y + job.start.y;
	unsigned int z = blockDim.z * blockIdx.z + threadIdx.z + job.start.z;

	if(x < job.end.x && y < job.end.y && z < job.end.z){
		REAL tmp1 = 1.0 / 3.0;
		REAL tmp2 = 2.0 / 3.0;

		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 0) = get_SoA(f , x,y,z , 0) = tmp1*get_SoA(fn , x,y,z , 0) + tmp2*( get_SoA(f , x,y,z , 0) + dt_d*get_SoA(du , x,y,z , 0) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 1) = get_SoA(f , x,y,z , 1) = tmp1*get_SoA(fn , x,y,z , 1) + tmp2*( get_SoA(f , x,y,z , 1) + dt_d*get_SoA(du , x,y,z , 1) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 2) = get_SoA(f , x,y,z , 2) = tmp1*get_SoA(fn , x,y,z , 2) + tmp2*( get_SoA(f , x,y,z , 2) + dt_d*get_SoA(du , x,y,z , 2) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 3) = get_SoA(f , x,y,z , 3) = tmp1*get_SoA(fn , x,y,z , 3) + tmp2*( get_SoA(f , x,y,z , 3) + dt_d*get_SoA(du , x,y,z , 3) );
		get_SoA_LAP(f_lap , x+LAP,y+LAP,z+LAP , 4) = get_SoA(f , x,y,z , 4) = tmp1*get_SoA(fn , x,y,z , 4) + tmp2*( get_SoA(f , x,y,z , 4) + dt_d*get_SoA(du , x,y,z , 4) );
	}
}

void OCFD_time_advance(int KRK)
{
	dim3 griddim , blockdim;
    cal_grid_block_dim(&griddim , &blockdim , BlockDimX , BlockDimY , BlockDimZ , nx,ny,nz);
    cudaJobPackage job( dim3(0,0,0) , dim3(nx,ny,nz) );

	switch (KRK)
	{
		case 1:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker1<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , job) ));
			break;
		}
		case 2:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker2<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , job) ));
			break;
		}
		case 3:
		{
			CUDA_LAUNCH(( OCFD_time_advance_ker3<<<griddim , blockdim>>>(*pf_d , *pfn_d , *pdu_d , *pf_lap_d , job) ));
			break;
		}
	}
}


#ifdef __cplusplus
}
#endif